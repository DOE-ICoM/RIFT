#include "hip/hip_runtime.h"
// Emacs Mode Line: -*- Mode:c++;-*-
// -------------------------------------------------------------
// file: GridSeries.cu
// -------------------------------------------------------------
// -------------------------------------------------------------
// Created August 24, 2023 by Perkins
// Last Change: 2024-12-30 11:22:55 d3g096
// -------------------------------------------------------------

#include <iostream>
#include <iomanip>
#include <sstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include "constants.h"
#include "io.h"
#include "grid.h"
#include "GridSeries.cuh"




// -------------------------------------------------------------
//  class GridSeries
// -------------------------------------------------------------

// -------------------------------------------------------------
// GridSeries:: constructors / destructor
// -------------------------------------------------------------
GridSeries::GridSeries(const std::string& basename,
                       const double& scale,
                       const int& deltat,
                       const double& tmax,
                       const struct GridConfig& gc,
                       double *dev_buf)
  : p_gc(gc),
    p_basename(basename),
    p_scale(scale),
    p_buffer(new double[gc.b_nx*gc.b_ny]()),
    p_int_buffer(new double[gc.h_nx*gc.h_ny]()),
    p_in_time(-9999.0), p_in_dt(deltat), p_max_time(tmax),
    p_current_dev(dev_buf),
    p_external(true), 
    p_done(false),
    p_allow_nodata(false),
    p_current_dev_init(true)
{
  if (p_current_dev == NULL) {
    
    // warning: global variables
    // Call SetDeviceConstants() first
    size_t width  = (GridDim.x * BlockDim.x) * sizeof(double);
    size_t height = (GridDim.y * BlockDim.y);
    
    checkCudaErrors(hipMallocPitch((void**)&p_current_dev, &pitch, width, height));
    p_init_dev();

    p_external = true;
  }
}

GridSeries::~GridSeries(void)
{
  if (!p_external) 
    hipFree(p_current_dev);
}

// -------------------------------------------------------------
// GridSeries::p_grid_name
// -------------------------------------------------------------
std::string
GridSeries::p_grid_name(const int& index) const
{
  std::stringstream hyetograph_file_ss;

  hyetograph_file_ss
    << p_basename << "-"
    << std::setw(3) << std::setfill('0') << index
    << ".txt";
  return hyetograph_file_ss.str();
}

// -------------------------------------------------------------
// GridSeries::p_interp
// -------------------------------------------------------------
void
GridSeries::p_interp(void)
{
  std::uninitialized_fill(p_int_buffer.get(),
                          p_int_buffer.get() + p_gc.h_nx*p_gc.h_ny,
                          (p_allow_nodata ? p_gc.nodata : 0.0));

  if (!p_current_dev_init) p_init_dev();

  for (int j = 2; j < p_gc.h_ny - 2; j++) {
    for (int i = 2; i < p_gc.h_nx - 2; i++) {
      int jt = j - 2, it = i - 2;
      
      int nnd(0);
      double vsum(0.0);
      for (int ni = 0; ni < 2; ++ni) {
        for (int nj = 0; nj < 2; ++nj) {
          int idx = (jt + nj) * p_gc.b_nx + (it + ni);
          if (p_buffer[idx] != p_gc.nodata) {
            nnd++;
            vsum += p_buffer[idx];
          }
        }
      }

      int index(j*p_gc.h_nx+i);
      if (nnd > 2) {
        p_int_buffer[index] = vsum/((double)nnd);
        p_int_buffer[index] *= p_scale;
      } 
    }
  }
}

// -------------------------------------------------------------
// GridSeries::p_read_grid
// -------------------------------------------------------------
void
GridSeries::p_read_grid(const int& index)
{
  std::string fname(p_grid_name(index));
  
  SetOriginalGrid(p_buffer.get(), fname, p_gc, p_allow_nodata);

  std::cout << "Reading from " << fname << " ..." << std::endl;

  this->p_interp();

}

// -------------------------------------------------------------
// GridSeries::p_update
// -------------------------------------------------------------
void
GridSeries::p_update(const double& t)
{
  bool sendit(false);
  int index;
  
  if (p_in_time < 0.0) {
    p_in_time = t;
    if (p_in_time < p_max_time) {
      index = trunc(p_in_time/p_in_dt);
      p_read_grid(index);
      sendit = true;
    }
  }

  // after the maximum time is reached, just fill w/ zeroes
  
  if (t >= p_max_time) {
    if (!p_done) {
      std::uninitialized_fill(p_int_buffer.get(),
                              p_int_buffer.get() + p_gc.h_nx*p_gc.h_ny,
                              0.0);
      sendit = true;
    }
    p_done = true;
  } else if (t >= (p_in_time + p_in_dt)) {
    p_in_time += p_in_dt;
    index = trunc(p_in_time/p_in_dt);
    p_read_grid(index);
    sendit = true;
  }

  if (sendit) {
    this->p_copy_to_dev();
  } 
}

// -------------------------------------------------------------
// GridSeries::p_sum
// -------------------------------------------------------------
double
GridSeries::p_sum(void) const
{
  // FIXME: do this on the device

  // return (ReduceSumGrid(p_current_dev));

  std::unique_ptr<double[]> tmp(new double[p_gc.h_nx*p_gc.h_ny]);
  checkCudaErrors(hipMemcpy2D(tmp.get(), p_gc.h_nx*sizeof(double), p_current_dev,
                               pitch, p_gc.h_nx*sizeof(double), p_gc.h_ny, DtoH));

  double result(0.0);

  for (int j = 2; j < p_gc.h_ny - 2; j++) {
    for (int i = 2; i < p_gc.h_nx - 2; i++) {
      if (tmp[j*p_gc.h_nx+i] != p_gc.nodata) {
        result += tmp[j*p_gc.h_nx+i];
      }
    }
  }

  return result;
}

// -------------------------------------------------------------
// GridSeries::p_init_dev
// -------------------------------------------------------------
void
GridSeries::p_init_dev()
{
  FillGrid(p_current_dev, (p_allow_nodata ? p_gc.nodata : 0.0));
  p_current_dev_init = true;
}

// -------------------------------------------------------------
// GridSeries::p_copy_to_dev
// -------------------------------------------------------------
void
GridSeries::p_copy_to_dev()
{
  checkCudaErrors(hipMemcpy2D(p_current_dev, pitch, &(p_int_buffer[0]),
                               p_gc.h_nx*sizeof(double), p_gc.h_nx*sizeof(double),
                               p_gc.h_ny, HtoD));
}

// -------------------------------------------------------------
//  class HyetographGridSeries
// -------------------------------------------------------------

const double HyetographGridSeries::scale(1.0/3600.0/1000.0);

// -------------------------------------------------------------
// HyetographGridSeries:: constructors / destructor
// -------------------------------------------------------------
HyetographGridSeries::HyetographGridSeries(const std::string& basename,
                                           const int& deltat,
                                           const double& tmax,
                                           const struct GridConfig& gc,
                                           double *dev_buf)
  : GridSeries(basename,
               HyetographGridSeries::scale, // converts mm/hr to m/s
               deltat, tmax, gc, dev_buf),
    p_sum_cache(0.0)
    
{}

HyetographGridSeries::~HyetographGridSeries(void)
{}

// -------------------------------------------------------------
// HyetographGridSeries::p_update
// -------------------------------------------------------------
void
HyetographGridSeries::p_update(const double& t)
{
  int index;
  
  if (p_in_time < 0.0) {
    GridSeries::p_update(t);
  }

  // rainfall between t and t + p_in_dt should be set from the map at
  // t + p_in_dt

  // Also, ince the hydrograph does not change over the input times,
  // the sum can be computed and cached
  if (t >= p_max_time) {
    GridSeries::p_update(t);
  } else if (t > (p_in_time)) {
    p_in_time += p_in_dt;
    index = trunc(p_in_time/p_in_dt);
    p_read_grid(index);
    this->p_copy_to_dev();
    p_sum_cache = GridSeries::p_sum();
  }
}

// -------------------------------------------------------------
// HyetographGridSeries::p_sum
// -------------------------------------------------------------
double
HyetographGridSeries::p_sum(void) const
{
  return p_sum_cache;
}

// -------------------------------------------------------------
//  class InterpolatedGridSeries
// -------------------------------------------------------------

// -------------------------------------------------------------
// InterpolatedGridSeries:: constructors / destructor
// -------------------------------------------------------------
InterpolatedGridSeries::InterpolatedGridSeries(const std::string& basename,
                                               const double& scale,
                                               const int& deltat,
                                               const double& tmax,
                                               const struct GridConfig& gc,
                                               double *dev_buf)
  : GridSeries(basename, scale, deltat, tmax, gc, dev_buf)
{
    // warning: global variables
    // Call SetDeviceConstants() first
    size_t width  = (GridDim.x * BlockDim.x) * sizeof(double);
    size_t height = (GridDim.y * BlockDim.y);
    
    checkCudaErrors(hipMallocPitch((void**)&p_t0_dev, &pitch, width, height));
    checkCudaErrors(hipMallocPitch((void**)&p_t1_dev, &pitch, width, height));
}

InterpolatedGridSeries::~InterpolatedGridSeries(void)
{}

// -------------------------------------------------------------
// InterpolatedGridSeries::p_update
// -------------------------------------------------------------
void
InterpolatedGridSeries::p_update(const double& t)
{
  int index;

  if (!p_current_dev_init) p_init_dev();
  
  if (p_in_time < 0.0) {
    
      p_in_time = t;
      index = trunc(p_in_time/p_in_dt);
      p_read_grid(index);

      checkCudaErrors(hipMemcpy2D(p_t0_dev, pitch, &(p_int_buffer[0]),
                                 p_gc.h_nx*sizeof(double), p_gc.h_nx*sizeof(double),
                                 p_gc.h_ny, HtoD));
      
      p_in_time = index*p_in_dt;
      p_next_time = p_in_time + p_in_dt;
      p_read_grid(index + 1);

      checkCudaErrors(hipMemcpy2D(p_t1_dev, pitch, &(p_int_buffer[0]),
                                 p_gc.h_nx*sizeof(double), p_gc.h_nx*sizeof(double),
                                 p_gc.h_ny, HtoD));

  } else if (t >= p_max_time) {

    if (!p_done) {
    
      p_in_time = p_next_time;
      p_next_time = p_in_time + p_in_dt;

      checkCudaErrors(hipMemcpy2D(p_t0_dev, pitch, p_t1_dev, pitch,
                                   p_gc.h_nx*sizeof(double), p_gc.h_ny,
                                   hipMemcpyDeviceToDevice));
      
    }
    p_done = true;
      
  } else if (t >= p_next_time) {

    checkCudaErrors(hipMemcpy2D(p_t0_dev, pitch, p_t1_dev, pitch,
                                 p_gc.h_nx*sizeof(double), p_gc.h_ny,
                                 hipMemcpyDeviceToDevice));
    
    p_in_time = p_next_time;
    p_next_time = p_in_time + p_in_dt;
    index = trunc(p_next_time/p_in_dt);
    p_read_grid(index);

    checkCudaErrors(hipMemcpy2D(p_t1_dev, pitch, &(p_int_buffer[0]),
                                 p_gc.h_nx*sizeof(double), p_gc.h_nx*sizeof(double),
                                 p_gc.h_ny, HtoD));

  }

  double factor((t - p_in_time)/(p_next_time - p_in_time));

  InterpGrid(factor, p_t0_dev, p_t1_dev, p_current_dev);
  
}  

// -------------------------------------------------------------
// InterpolatedGridSeries::p_init_dev
// -------------------------------------------------------------
void
InterpolatedGridSeries::p_init_dev()
{
  GridSeries::p_init_dev();
  FillGrid(p_t0_dev, (p_allow_nodata ? p_gc.nodata : 0.0));
  FillGrid(p_t1_dev, (p_allow_nodata ? p_gc.nodata : 0.0));
}

// -------------------------------------------------------------
// InterpolatedGridSeries::p_copy_to_dev
// -------------------------------------------------------------
void
InterpolatedGridSeries::p_copy_to_dev(void)
{
}  

